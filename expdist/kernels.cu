#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "kernels.cuh"

extern "C"
__global__ void
ExpDist(const double *A, const double *B,
                 int m, int n, const double *scale_A, const double *scale_B, double *cross_term) {

    //2-dimensional with double precision
    ExpDist_tiled<double, 2>(A, B, m, n, scale_A, scale_B, cross_term);

}

extern "C"
__global__ void
ExpDist3D(const double *A, const double *B,
                 int m, int n, const double *scale_A, const double *scale_B, double *cross_term) {

    //3-dimensional with double precision
    ExpDist3D_tiled<double, 3>(A, B, m, n, scale_A, scale_B, cross_term);

}

extern "C"
__global__ void
ExpDist_column(const double *A, const double *B,
                 int m, int n, const double *scale_A, const double *scale_B, double *cross_term) {

    //2-dimensional with double precision
    //ExpDist_tiled_column<double, 2>(A, B, m, n, scale_A, scale_B, cross_term);

}

extern "C"
__global__ void
ExpDist_column3D(const double *A, const double *B,
                 int m, int n, const double *scale_A, const double *scale_B, double *cross_term) {


    return;
}

/*
 * Reduce the per thread block cross terms computed in the GaussTransform kernel to single value
 *
 * This kernel is designed to run as single-thread block, because the number of terms to reduce is
 * of size n or m, which is expected to be around 2000 or so. The number of items to reduce
 * is passed as the last argument 'nblocks', which corresponds to the number of thread blocks used
 * by the first kernel.
 */
extern "C"
__global__ void reduce_cross_term(double *output, double *d_cross_term, int m, int n, int nblocks) {

    int tx = threadIdx.x;
    // Specialize BlockReduce for a 1D block of block_size threads on type double
    typedef hipcub::BlockReduce<double, block_size> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    double cross_term = 0.0;
    for (int i=tx; i<nblocks; i+=block_size) {
        cross_term += d_cross_term[i];
    }

    //reduce to single value within thread block
    cross_term = BlockReduce(temp_storage).Sum(cross_term);

    //thread 0 writes output
    if (tx == 0) {
        output[0] = cross_term;
    }

}


extern "C"
__global__ void rotate_scales_double(double *rotated_scales, const int n, const double *scale_B, double const * rotation_matrix, double const * rotation_matrix_transposed) {

    int x = blockIdx.x * block_size_x + threadIdx.x;

    if (x < n) {
        rotate_scale(rotated_scales, rotation_matrix, rotation_matrix_transposed, n, x, scale_B);
    }

}

extern "C"
__global__ void rotate_B_double(double *rotated_B, const int n, const double *B, double const * rotation_matrix) {

    int x = blockIdx.x * block_size_x + threadIdx.x;

    if (x < n) {
        rotate_B_point(rotated_B, rotation_matrix, x, B);
    }

}
